#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "common.h"

#define NUM_THREADS 256

extern double size;
//
//  benchmarking program
//

__global__ void clear_binSize(int * d_binSize, int numBins) {
    // Get thread (particle) ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid >= numBins * numBins) return;

    d_binSize[tid] = 0;

}

__global__ void generate_bins(particle_t * particles, particle_t * * bins, int * binSize, int n, int pitch) {
    // Get thread (particle) ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid >= n) return;

    int currBinIndex = particles[tid].binIndex;
    int index = atomicAdd(&binSize[currBinIndex], 1);

    bins[currBinIndex * pitch + index] = &particles[tid];

}

__device__ void apply_force_gpu(particle_t * particle, particle_t neighbor) {
    double dx = neighbor.x - particle->x;
    double dy = neighbor.y - particle->y;
    double r2 = dx * dx + dy * dy;
    if (r2 > cutoff * cutoff)
        return;
    //r2 = fmax( r2, min_r*min_r );
    r2 = (r2 > min_r * min_r) ? r2 : min_r*min_r;
    double r = sqrt(r2);

    //
    //  very simple short-range repulsive force
    //
    double coef = (1 - cutoff / r) / r2 / mass;
    particle->ax += coef * dx;
    particle->ay += coef * dy;
}

__device__ void bin_apply_force_gpu(particle_t * particle, particle_t * * bin, int length) {
   for (int i = 0; i < length; i++)
       apply_force_gpu(particle, *bin[i]);
    //particle->newIndex = bin[i]->index + 10; //ok
}

__global__ void compute_forces_gpu(particle_t * particles, particle_t * * bins, int * binSize, int n, int numBins, int pitch) {
    // Get thread (particle) ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid >= n) return;

    particle_t * currParticle = &particles[tid];
    int currBinIndex = currParticle->binIndex;

    //handle first bin row -------------------------------------------------
    //
    //1st column special case (j=0))
    //            
    if (currBinIndex == 0) {
        //compare with self
        bin_apply_force_gpu(currParticle, &bins[0], binSize[0]);
        //compare with right
        bin_apply_force_gpu(currParticle, &bins[pitch], binSize[1]);
        //compare low
        bin_apply_force_gpu(currParticle, &bins[numBins * pitch], binSize[numBins]);
        //compare with low-right
        bin_apply_force_gpu(currParticle, &bins[(numBins + 1) * pitch], binSize[numBins + 1]);
    }//
        //jth bin column for j=1 to numBins-2
        //
    else if (currBinIndex < numBins - 1) {
        //compare with left
        bin_apply_force_gpu(currParticle, &bins[(currBinIndex - 1) * pitch], binSize[currBinIndex - 1]);
        //compare with self
        bin_apply_force_gpu(currParticle, &bins[currBinIndex * pitch], binSize[currBinIndex]);
        //compare with right
        bin_apply_force_gpu(currParticle, &bins[(currBinIndex + 1) * pitch], binSize[currBinIndex + 1]);
        //compare with low-left
        bin_apply_force_gpu(currParticle, &bins[(numBins + currBinIndex - 1) * pitch], binSize[numBins + currBinIndex - 1]);
        //compare with low
        bin_apply_force_gpu(currParticle, &bins[(numBins + currBinIndex) * pitch], binSize[numBins + currBinIndex]);
        //compare with low-right    
        bin_apply_force_gpu(currParticle, &bins[(numBins + currBinIndex + 1) * pitch], binSize[numBins + currBinIndex + 1]);
    }//
        //last bin column (j = numBins-1)
        //
    else if (currBinIndex == numBins - 1) {
        //compare with left
        bin_apply_force_gpu(currParticle, &bins[(numBins - 2) * pitch], binSize[numBins - 2]);
        //compare with self
        bin_apply_force_gpu(currParticle, &bins[(numBins - 1) * pitch], binSize[numBins - 1]);
        //compare with low-left
        bin_apply_force_gpu(currParticle, &bins[(numBins + numBins - 2) * pitch], binSize[numBins + numBins - 2]);
        //compare with low
        bin_apply_force_gpu(currParticle, &bins[(numBins + numBins - 1) * pitch], binSize[numBins + numBins - 1]);
    }//
        //ith bin rows for i = 1 to numBins-2
        //
  else if (currBinIndex < (numBins - 1) * numBins) {
        //
        //1st column special case (j=0)
        //
        if (currBinIndex % numBins == 0) {
            //compare with up
            bin_apply_force_gpu(currParticle, &bins[(currBinIndex - numBins) * pitch], binSize[currBinIndex - numBins]);
            //compare with up-right
            bin_apply_force_gpu(currParticle, &bins[(currBinIndex - numBins + 1) * pitch], binSize[currBinIndex - numBins + 1]);
            //compare with self
            bin_apply_force_gpu(currParticle, &bins[currBinIndex * pitch], binSize[currBinIndex]);
            //compare with right
            bin_apply_force_gpu(currParticle, &bins[(currBinIndex + 1) * pitch], binSize[currBinIndex + 1]);
            //compare with low
            bin_apply_force_gpu(currParticle, &bins[(currBinIndex + numBins) * pitch], binSize[currBinIndex + numBins]);
            //compare with low-right
            bin_apply_force_gpu(currParticle, &bins[(currBinIndex + numBins + 1) * pitch], binSize[currBinIndex + numBins + 1]);
        }//
            //jth column for j=1 to numBins-2
            //
        else if (currBinIndex % numBins != (numBins - 1)) {
            //compare with up-left
            bin_apply_force_gpu(currParticle, &bins[(currBinIndex - numBins - 1) * pitch], binSize[currBinIndex - numBins - 1]);
            //compare with up
            bin_apply_force_gpu(currParticle, &bins[(currBinIndex - numBins) * pitch], binSize[currBinIndex - numBins]);
            //compare with up-right
            bin_apply_force_gpu(currParticle, &bins[(currBinIndex - numBins + 1) * pitch], binSize[currBinIndex - numBins + 1]);
            //compare with left
            bin_apply_force_gpu(currParticle, &bins[(currBinIndex - 1) * pitch], binSize[currBinIndex - 1]);
            //compare with self
            bin_apply_force_gpu(currParticle, &bins[(currBinIndex) * pitch], binSize[currBinIndex]);
            //compare with right
            bin_apply_force_gpu(currParticle, &bins[(currBinIndex + 1) * pitch], binSize[currBinIndex + 1]);
            //compare with low-left
            bin_apply_force_gpu(currParticle, &bins[(currBinIndex + numBins - 1) * pitch], binSize[currBinIndex + numBins - 1]);
            //compare with low
            bin_apply_force_gpu(currParticle, &bins[(currBinIndex + numBins) * pitch], binSize[currBinIndex + numBins]);
            //compare with low-right
            bin_apply_force_gpu(currParticle, &bins[(currBinIndex + numBins + 1) * pitch], binSize[currBinIndex + numBins + 1]);
        }//
            //numBins-1 (last column) special case
            //
        else {
            //compare with up-left
            bin_apply_force_gpu(currParticle, &bins[(currBinIndex - numBins - 1) * pitch], binSize[currBinIndex - numBins - 1]);
            //compare with up
            bin_apply_force_gpu(currParticle, &bins[(currBinIndex - numBins) * pitch], binSize[currBinIndex - numBins]);
            //compare with left
            bin_apply_force_gpu(currParticle, &bins[(currBinIndex - 1) * pitch], binSize[currBinIndex - 1]);
            //compare with self
            bin_apply_force_gpu(currParticle, &bins[currBinIndex * pitch], binSize[currBinIndex]);
            //compare with low-left
            bin_apply_force_gpu(currParticle, &bins[(currBinIndex + numBins - 1) * pitch], binSize[currBinIndex + numBins - 1]);
            //compare with low
            bin_apply_force_gpu(currParticle, &bins[(currBinIndex + numBins) * pitch], binSize[currBinIndex + numBins]);
        }
    }//
        //numBins-1 (last row)
        //
     else if (currBinIndex >= (numBins - 1) * numBins && currBinIndex < numBins * numBins) {
        //
        //1st column special case (j=0)
        //
        if (currBinIndex % numBins == 0) {
            //compare up
            bin_apply_force_gpu(currParticle, &bins[(currBinIndex - numBins) * pitch], binSize[currBinIndex - numBins]);
            //compare up-right
            bin_apply_force_gpu(currParticle, &bins[(currBinIndex - numBins + 1) * pitch], binSize[currBinIndex - numBins + 1]);
            //compare self
            bin_apply_force_gpu(currParticle, &bins[currBinIndex * pitch], binSize[currBinIndex]);
            //compare right
            bin_apply_force_gpu(currParticle, &bins[(currBinIndex + 1) * pitch], binSize[currBinIndex + 1]);
        }//
            //jth column for j=1 to numBins-2
            //
        else if (currBinIndex % numBins != (numBins - 1)) {
            //compare with up-left
            bin_apply_force_gpu(currParticle, &bins[(currBinIndex - numBins - 1) * pitch], binSize[currBinIndex - numBins - 1]);
            //compare with up
            bin_apply_force_gpu(currParticle, &bins[(currBinIndex - numBins) * pitch], binSize[currBinIndex - numBins]);
            //compare with up-right
            bin_apply_force_gpu(currParticle, &bins[(currBinIndex - numBins + 1) * pitch], binSize[currBinIndex - numBins + 1]);
            //compare with left
            bin_apply_force_gpu(currParticle, &bins[(currBinIndex - 1) * pitch], binSize[currBinIndex - 1]);
            //compare with self
            bin_apply_force_gpu(currParticle, &bins[currBinIndex * pitch], binSize[currBinIndex]);
            //compare with right
            bin_apply_force_gpu(currParticle, &bins[(currBinIndex + 1) * pitch], binSize[currBinIndex + 1]);
        }//
            //numBins-1 (last column) special case
            //
        else {
            //compare with up-left
            bin_apply_force_gpu(currParticle, &bins[(currBinIndex + numBins - 1) * pitch], binSize[currBinIndex + numBins - 1]);
            //compare with up
            bin_apply_force_gpu(currParticle, &bins[(currBinIndex + numBins) * pitch], binSize[currBinIndex + numBins]);
            //compare with left
            bin_apply_force_gpu(currParticle, &bins[(currBinIndex - 1) * pitch], binSize[currBinIndex - 1]);
            //compare with self
            bin_apply_force_gpu(currParticle, &bins[currBinIndex * pitch], binSize[currBinIndex]);
        }
    }
}

__global__ void move_gpu(particle_t * particles, int n, double size, int numBins, double binLength) {

    // Get thread (particle) ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= n) return;

    particle_t * p = &particles[tid];
    //
    //  slightly simplified Velocity Verlet integration
    //  conserves energy better than explicit Euler method
    //
    p->vx += p->ax * dt;
    p->vy += p->ay * dt;
    p->x += p->vx * dt;
    p->y += p->vy * dt;

    //
    //  bounce from walls
    //
    while (p->x < 0 || p->x > size) {
        p->x = p->x < 0 ? -(p->x) : 2 * size - p->x;
        p->vx = -(p->vx);
    }
    while (p->y < 0 || p->y > size) {
        p->y = p->y < 0 ? -(p->y) : 2 * size - p->y;
        p->vy = -(p->vy);
    }

    p->ax = p->ay = 0;
    p->binIndex = ((int) (p->x / binLength)) + ((int) (p->y / binLength)) * numBins;
}

int main(int argc, char **argv) {
    // This takes a few seconds to initialize the runtime
    hipDeviceSynchronize();

    if (find_option(argc, argv, "-h") >= 0) {
        printf("Options:\n");
        printf("-h to see this help\n");
        printf("-n <int> to set the number of particles\n");
        printf("-o <filename> to specify the output file name\n");
        printf("-s <filename> to specify the summary output file name\n");
        return 0;
    }

    int n = read_int(argc, argv, "-n", 1000);

    set_size(n);
    double size = get_size();
    double binLength = 2 * cutoff;
    int numBins = ceil(size / binLength);

    char *savename = read_string(argc, argv, "-o", NULL);
    char *sumname = read_string(argc, argv, "-s", NULL);

    FILE *fsave = savename ? fopen(savename, "w") : NULL;
    FILE *fsum = sumname ? fopen(sumname, "a") : NULL;
    setbuf(stdout, NULL);

    particle_t *particles = (particle_t*) malloc(n * sizeof (particle_t));
    // GPU particle data structure
    particle_t * d_particles;
    hipMalloc((void **) &d_particles, n * sizeof (particle_t));

    particle_t * * d_bins;
    int perBin = 10;
    hipMalloc((void **) &d_bins, perBin * numBins * numBins * sizeof (particle_t *));

    int * d_binSize;
    hipMalloc((void **) &d_binSize, numBins * numBins * sizeof (int));

    init_particles(n, particles);

    hipDeviceSynchronize();
    double copy_time = read_timer();

    // Copy the particles to the GPU
    hipMemcpy(d_particles, particles, n * sizeof (particle_t), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    copy_time = read_timer() - copy_time;

    //
    //  simulate a number of time steps
    //
    hipDeviceSynchronize();
    double simulation_time = read_timer();
    int blks = (n + NUM_THREADS - 1) / NUM_THREADS;
    int binBlks = (numBins * numBins + NUM_THREADS - 1) / NUM_THREADS;
    for (int step = 0; step < NSTEPS; step++) {

        //
        // clear bins
        //
        clear_binSize << <binBlks, NUM_THREADS >> > (d_binSize, numBins);

        //
        //GENERATE BIN
        //
        generate_bins << < blks, NUM_THREADS >> > (d_particles, d_bins, d_binSize, n, perBin);
         
        //
        //  compute forces
        //
        compute_forces_gpu << <blks, NUM_THREADS >> > (d_particles, d_bins, d_binSize, n, numBins, perBin);
        
        //
        //  move particles
        //        
        move_gpu << < blks, NUM_THREADS >> > (d_particles, n, size, numBins, binLength);
        
        //
        //  save if necessary
        //
        if (fsave && (step % SAVEFREQ) == 0) {
            // Copy the particles back to the CPU
            hipMemcpy(particles, d_particles, n * sizeof (particle_t), hipMemcpyDeviceToHost);
            save(fsave, n, particles);
        }
    }
    hipDeviceSynchronize();

    simulation_time = read_timer() - simulation_time;

    printf("CPU-GPU copy time = %g seconds\n", copy_time);
    printf("n = %d, simulation time = %g seconds\n", n, simulation_time);

    if (fsum)
        fprintf(fsum, "%d %lf \n", n, simulation_time);

    if (fsum)
        fclose(fsum);
    free(particles);
    hipFree(d_particles);
    hipFree(d_bins);
    hipFree(d_binSize);
    if (fsave)
        fclose(fsave);

    return 0;
}
